// nvcc -std=c++11 GPU_main.cu -o GPU_main -lrealsense2 -lboost_iostreams -lboost_system -lboost_filesystem `pkg-config opencv --cflags --libs` -lpthread -Wno-deprecated-gpu-targets

#include <opencv2/opencv.hpp>

#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <unistd.h>
#include <mutex>
#include <thread>
#include <atomic>
#include <chrono>
#include <time.h>

#include <boost/tuple/tuple.hpp>

#include "../include/Voxel.cuh"
#include "../include/Logging.hpp"




int main(int argc, char const *argv[])
{
	std::atomic_bool alive {true};

    hipDeviceReset();
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 10ull*1024ull*1024ull);
    cudaThreadSetLimit (hipLimitMallocHeapSize, 2048ull*1024ull*1024ull);

    /* Map Front End */
    Map_FE * F = new GPU_FE();

    /* Camera Initialization */
    Camera C;
    Bool_Init bC = C.Init();
    if (bC.t265 && bC.d435)
        std::cout << "Cameras initialized\n";
    else 
        std::cout << "Atleast one camera is not connected\n";

    /* Logger Initialization */
    Logger L;
    L.Init();

    /* Thread for checking exit condition */

    std::thread exit_check([&]() {
        while (alive) {
            if (std::cin.get() == ' ') {
                cv::destroyAllWindows();
                alive = false;
            }
        }
    });

    /* Thread for receiving frames and storing them as video and csv files */ 

    std::thread rxFrame([&]() {
        while (alive) {
            auto sleep_start = std::chrono::high_resolution_clock::now();

			auto tframe = C.pipelines[0].wait_for_frames();
			auto dframe = C.pipelines[1].wait_for_frames();

			auto t = tframe.first_or_default(RS2_STREAM_POSE);
			auto d = dframe.get_depth_frame();

			if (!t || !d)
				continue;

			C.t_queue.enqueue(tframe);
			C.d_queue.enqueue(dframe);

            // sleep for remaining time
            auto time_sleep = std::chrono::high_resolution_clock::now() - sleep_start;
            double time_s = std::chrono::duration_cast<std::chrono::milliseconds>(time_sleep).count();
            if ((1000.0/INPUT_RATE)-time_s > 0){
                usleep((1000.0/INPUT_RATE-time_s) * 1000);
            }
            // std::cout << time_s << "\n";
		}
    });

//bool en = false;
    rs2::frameset t_frameset, d_frameset;
    auto start = std::chrono::high_resolution_clock::now();

    while (alive) {
    	C.t_queue.poll_for_frame(&t_frameset);
    	C.d_queue.poll_for_frame(&d_frameset);

    	if (t_frameset && d_frameset) {
    		auto depthFrame = d_frameset.get_depth_frame();
    		auto poseFrame  = t_frameset.first_or_default(RS2_STREAM_POSE);

    		cv::Mat depth(cv::Size(w, h), CV_16UC1, (void *)depthFrame.get_data(), cv::Mat::AUTO_STEP);
    		auto pose = poseFrame.as<rs2::pose_frame>().get_pose_data();

            /* update global map */
            //if (!en) {
            F->Update (C, pose, depth);
            //en = true;
            //}
            /*                   */

    		auto elapsed = std::chrono::high_resolution_clock::now() - start;
            float microseconds = std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
            std::cout << microseconds << "\n";

            L.Log(&C, &pose, &depth);

    	}

    	start = std::chrono::high_resolution_clock::now();

    }

    rxFrame.join();

    L.Close(&C, F);

    std::cout << "Program terminated sucessfully\n";
	return 0;
	
}
